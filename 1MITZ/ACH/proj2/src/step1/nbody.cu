#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xpavli78
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"
#include <iostream>
#include <random>
#include <exception>
#include <string>
#include <cstdio>

using namespace std;

__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
		float r;


		for (int j = 0; j < N; j++) {
			float dx = p_in.pos[i].x - p_in.pos[j].x;
			float dy = p_in.pos[i].y - p_in.pos[j].y;
			float dz = p_in.pos[i].z - p_in.pos[j].z;

			r = sqrtf(dx*dx + dy*dy + dz*dz);

			float dw = p_in.pos[i].w - p_in.pos[j].w ;
			float totalw = p_in.pos[i].w + p_in.pos[j].w ;
			float r3 = r * r * r;
			float velocity = (G * dt * p_in.pos[j].w * -1.0f) / r3;

			Fx += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * p_in.pos[j].w * p_in.vel[j].x + dw * p_in.vel[i].x) / totalw) - p_in.vel[i].x : 0.0f;
			Fy += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * p_in.pos[j].w * p_in.vel[j].y + dw * p_in.vel[i].y) / totalw) - p_in.vel[i].y : 0.0f;
			Fz += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * p_in.pos[j].w * p_in.vel[j].z + dw * p_in.vel[i].z) / totalw) - p_in.vel[i].z : 0.0f;



			Fx += (r > COLLISION_DISTANCE) ? dx * velocity : 0.0f;
			Fy += (r > COLLISION_DISTANCE) ? dy * velocity : 0.0f;
			Fz += (r > COLLISION_DISTANCE) ? dz * velocity : 0.0f;

		}

		Fx += p_in.vel[i].x;
		Fy += p_in.vel[i].y;
		Fz += p_in.vel[i].z;



		p_out.vel[i].x =  Fx;
		p_out.vel[i].y =  Fy;
		p_out.vel[i].z =  Fz;

		p_out.pos[i].x = (Fx * dt) + p_in.pos[i].x;
		p_out.pos[i].y = (Fy * dt) + p_in.pos[i].y;
		p_out.pos[i].z = (Fz * dt) + p_in.pos[i].z;

		//printf("%f %f %f %f %f %f %f %f xaxa\n", p_out.pos[i].x,p_out.pos[i].y,p_out.pos[i].z,p_out.pos[i].w,p_out.vel[i].x,p_out.vel[i].y,p_out.vel[i].z,p_out.vel[i].w);

	}
}

/*
__global__ void calculate_velocity(const t_particles p_in, t_particles p_out, const int N, const float dt)
{

	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		float Fx = p_in.vel[i].x; float Fy = p_in.vel[i].y; float Fz = p_in.vel[i].z;
		float r;

		for (int j = 0; j < N; j++) {
			float dx = p_in.pos[i].x - p_in.pos[j].x;
			float dy = p_in.pos[i].y - p_in.pos[j].y;
			float dz = p_in.pos[i].z - p_in.pos[j].z;

			r = sqrtf(dx*dx + dy*dy + dz*dz);
			float r3 = r * r * r;

			float velocity = (G * dt * p_in.pos[j].w -1.0f) / r3;

			Fx += (r > COLLISION_DISTANCE) ? dx * velocity : 0.0f;
			Fy += (r > COLLISION_DISTANCE) ? dy * velocity : 0.0f;
			Fz += (r > COLLISION_DISTANCE) ? dz * velocity : 0.0f;

			float dw = p_in.pos[i].w - p_in.pos[j].w;
			float totalw = p_in.pos[i].w + p_in.pos[j].w;

			Fx += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2 * p_in.pos[j].w * p_in.vel[j].x + dw * p_in.vel[i].x) / totalw) - p_in.vel[i].x : 0.0f;
			Fy += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2 * p_in.pos[j].w * p_in.vel[j].y + dw * p_in.vel[i].y) / totalw) - p_in.vel[i].y : 0.0f;
			Fz += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2 * p_in.pos[j].w * p_in.vel[j].z + dw * p_in.vel[i].z) / totalw) - p_in.vel[i].z : 0.0f;
		}

		p_out.vel[i].x = Fx;
		p_out.vel[i].y = Fy;
		p_out.vel[i].z = Fz;

		p_out.pos[i].x = Fx * dt + p_in.pos[i].x;
		p_out.pos[i].y = Fy * dt + p_in.pos[i].y;
		p_out.pos[i].z = Fz * dt + p_in.pos[i].z;
	}

}
 */
/*
__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {

		float Fx = p_in.vel[i].x; float Fy = p_in.vel[i].y; float Fz = p_in.vel[i].z;
		float r;

		for (int j = 0; j < N; j++) {

			float dx = p_in.pos[i].x - p_in.pos[j].x;
			float dy = p_in.pos[i].y - p_in.pos[j].y;
			float dz = p_in.pos[i].z - p_in.pos[j].z;

			float dw = p_in.pos[i].w - p_in.pos[j].w ;
			float totalw = p_in.pos[i].w + p_in.pos[j].w ;

			r = sqrtf(dx*dx + dy*dy + dz*dz);
			float r3 = r * r * r;
			float velocity = (G * dt * p_in.pos[j].w * -1.0f) / r3;

			Fx += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * p_in.pos[j].w * p_in.vel[j].x + dw * p_in.vel[i].x) / totalw) - p_in.vel[i].x : 0.0f;
			Fy += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * p_in.pos[j].w * p_in.vel[j].y + dw * p_in.vel[i].y) / totalw) - p_in.vel[i].y : 0.0f;
			Fz += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * p_in.pos[j].w * p_in.vel[j].z + dw * p_in.vel[i].z) / totalw) - p_in.vel[i].z : 0.0f;

			Fx += (r > COLLISION_DISTANCE) ? dx * velocity : 0.0f;
			Fy += (r > COLLISION_DISTANCE) ? dy * velocity : 0.0f;
			Fz += (r > COLLISION_DISTANCE) ? dz * velocity : 0.0f;
		}


		p_out.vel[i].x =  Fx;
		p_out.vel[i].y =  Fy;
		p_out.vel[i].z =  Fz;

		p_out.pos[i].x = (Fx * dt) + p_in.pos[i].x;
		p_out.pos[i].y = (Fy * dt) + p_in.pos[i].y;
		p_out.pos[i].z = (Fz * dt) + p_in.pos[i].z;

	}
	//printf("%f %f %f %f %f %f %f %f xaxa\n", p_out.pos[i].x,p_out.pos[i].y,p_out.pos[i].z,p_out.pos[i].w,p_out.vel[i].x,p_out.vel[i].y,p_out.vel[i].z,p_out.vel[i].w);

}
 */


/*
__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
		float r;

		float px = p_in.pos[i].x;
		float py = p_in.pos[i].y;
		float pz = p_in.pos[i].z;

		float pvx = p_in.vel[i].x;
		float pvy = p_in.vel[i].y;
		float pvz = p_in.vel[i].z;

		float pw = p_in.pos[i].w;

		for (int j = 0; j < N; j++) {
			float dx = px - p_in.pos[j].x;
			float dy = py - p_in.pos[j].y;
			float dz = pz - p_in.pos[j].z;

			float pwi = p_in.pos[j].w;

			r = sqrtf(dx*dx + dy*dy + dz*dz);

			if(r > 0.0f && r < COLLISION_DISTANCE)
			{
				float dw = pw - pwi ;
				float totalw = pw + pwi ;

				Fx += ((2.0f * pwi * p_in.vel[j].x + dw * pvx) / totalw) - pvx;
				Fy += ((2.0f * pwi * p_in.vel[j].y + dw * pvy) / totalw) - pvy;
				Fz += ((2.0f * pwi * p_in.vel[j].z + dw * pvz) / totalw) - pvz;

			}
			else
			{
				float r3 = r * r * r;
				float velocity = (G * dt * pwi * -1.0f) / r3;

				Fx += (r > COLLISION_DISTANCE) ? dx * velocity : 0.0f;
				Fy += (r > COLLISION_DISTANCE) ? dy * velocity : 0.0f;
				Fz += (r > COLLISION_DISTANCE) ? dz * velocity : 0.0f;
			}

		}

		Fx += pvx;
		Fy += pvy;
		Fz += pvz;



		p_out.vel[i].x =  Fx;
		p_out.vel[i].y =  Fy;
		p_out.vel[i].z =  Fz;

		p_out.pos[i].x = (Fx * dt) + px;
		p_out.pos[i].y = (Fy * dt) + py;
		p_out.pos[i].z = (Fz * dt) + pz;

		//printf("%f %f %f %f %f %f %f %f xaxa\n", p_out.pos[i].x,p_out.pos[i].y,p_out.pos[i].z,p_out.pos[i].w,p_out.vel[i].x,p_out.vel[i].y,p_out.vel[i].z,p_out.vel[i].w);

	}
}
 */
__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
	/* DOPLNTE: telo funkce pro nacitani castic */
	for (int i = 0; i < N; i++)
	{
		fscanf(fp, "%f %f %f %f %f %f %f \n",
				&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
				&p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
				&p.pos[i].w);
	}
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
	for (int i = 0; i < N; i++)
	{
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
				p.pos[i].x, p.pos[i].y, p.pos[i].z,
				p.vel[i].x, p.vel[i].y, p.vel[i].z,
				p.pos[i].w);
	}
	/* DOPLNTE: telo funkce pro ulozeni castic */
}
