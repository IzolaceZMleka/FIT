#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xpavli78
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <iostream>
#include <algorithm>

#include "nbody.h"

// Makro pre kontrolu chyb CUDA kniznice
#define cudaCheckError() __cudaCheckError( __FILE__, __LINE__ )

// Kontrola chyb CUDA
inline void __cudaCheckError( const char *file, const int line )
{
  using namespace std;
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
  {
    cerr<<"cudaCheckError() failed at "<<file<<":"<<line<<" : "<<hipGetErrorString( err )<<endl;
    exit( -1 );
  }
}

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

    // alokace pameti na CPU
    t_particles particles_cpu;
    size_t partArr = N*sizeof(float4);

    hipHostAlloc(&particles_cpu.pos, partArr, hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel, partArr, hipHostMallocDefault);
    cudaCheckError();

    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[5]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);

    t_particles particles_gpu1;
    t_particles particles_gpu2;

    /* DOPLNTE: alokaci pameti na GPU */
    hipMalloc(&particles_gpu1.pos, partArr);
    hipMalloc(&particles_gpu1.vel, partArr);
    hipMalloc(&particles_gpu2.pos, partArr);
    hipMalloc(&particles_gpu2.vel, partArr);
    cudaCheckError();

    /* DOPLNTE: kopirovani pameti na GPU */

    hipMemcpy(particles_gpu1.pos, particles_cpu.pos, partArr, hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu1.vel, particles_cpu.vel, partArr, hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu2.pos, particles_cpu.pos, partArr, hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu2.vel, particles_cpu.vel, partArr, hipMemcpyHostToDevice);
    cudaCheckError();

    /* DOPLNTE: vypocet velikosti vypocetniho gridu */
    size_t grid = (N + thr_blc - 1) / thr_blc;;


    // vypocet
    gettimeofday(&t1, 0);

    for (int s = 0; s < steps; ++s)
    {
        calculate_velocity<<<grid, thr_blc>>>(particles_gpu1, particles_gpu2, N, dt);
        swap(particles_gpu1, particles_gpu2);
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    /* DOPLNTE: koopirovani castic spatky na CPU */

    hipMemcpy(particles_cpu.pos,  particles_gpu1.pos,  partArr, hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel,  particles_gpu1.vel,  partArr, hipMemcpyDeviceToHost);

    cudaCheckError();

    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);

    /* dealokaci pameti na CPU */
	if(particles_cpu.pos)  hipFree(particles_cpu.pos);
	if(particles_cpu.vel)  hipFree(particles_cpu.vel);
	if(particles_gpu1.pos)  hipFree(particles_gpu1.pos);
	if(particles_gpu1.vel)  hipFree(particles_gpu1.vel);
	if(particles_gpu2.pos)  hipFree(particles_gpu2.pos);
	if(particles_gpu2.vel)  hipFree(particles_gpu2.vel);

    return 0;
}

