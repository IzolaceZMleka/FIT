#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xpavli78
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"
#include <iostream>
#include <random>
#include <exception>
#include <string>
#include <cstdio>

using namespace std;

__global__ void calculate_velocity(t_particles p_in, t_particles p_out, int N, float dt)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
        //sdílená paměť velikost 2x počet vláken * sizeof(float4)
	extern __shared__ float4 sharedP[];

	if (i < N) {
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
		float r;
                //soubezna iterace pres vsechna vlakna
		for(int k = 0; k < N/blockDim.x; k++)
		{
                        //zjisteni indexu v ramci globalni pameti
			int index = (threadIdx.x + k * blockDim.x);
                        //nahrani do sdílené paměti, index * 2, na index + 1 se nachází druhý float4
			sharedP[threadIdx.x*2] = p_in.pos[index];
			sharedP[threadIdx.x*2 + 1] = p_in.vel[index];
                        //synchronizace, aby nedošlo k přístupu k neincializovaným datům
			__syncthreads();
                        // prrubezny vypocet nad prave nahranym usekem odpovidajici poctu vlaken do globalni pameti
                        //vypocet jednotlivych vlaken, kde vsechny "druhe" castice jsou nahrazen jejich vyskytem v sdilene pameti
			for (int j = 0; j < blockDim.x; j++) {
				float dx = p_in.pos[i].x - sharedP[j*2].x;
				float dy = p_in.pos[i].y - sharedP[j*2].y;
				float dz = p_in.pos[i].z - sharedP[j*2].z;

				r = sqrtf(dx*dx + dy*dy + dz*dz);

				float dw = p_in.pos[i].w - sharedP[j*2].w ;
				float totalw = p_in.pos[i].w + sharedP[j*2].w ;
				float r3 = r * r * r;
				float velocity = (G * dt * sharedP[j*2].w * -1.0f) / r3;

				Fx += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * sharedP[j*2].w * sharedP[j*2 ].x + dw * p_in.vel[i].x) / totalw) - p_in.vel[i].x : 0.0f;
				Fy += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * sharedP[j*2].w * sharedP[j*2 ].y + dw * p_in.vel[i].y) / totalw) - p_in.vel[i].y : 0.0f;
				Fz += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2.0f * sharedP[j*2].w * sharedP[j*2 ].z + dw * p_in.vel[i].z) / totalw) - p_in.vel[i].z : 0.0f;

				Fx += (r > COLLISION_DISTANCE) ? dx * velocity : 0.0f;
				Fy += (r > COLLISION_DISTANCE) ? dy * velocity : 0.0f;
				Fz += (r > COLLISION_DISTANCE) ? dz * velocity : 0.0f;

			}
                        //synchronizace po ukonceni vypoctu, aby nedoslo k drivejsimu nahrani do sdilene pameti
			__syncthreads();

		}
                //aktualizace pozice a rychlosti castice 
		Fx += p_in.vel[i].x;
		Fy += p_in.vel[i].y;
		Fz += p_in.vel[i].z;

		p_out.vel[i].x =  Fx;
		p_out.vel[i].y =  Fy;
		p_out.vel[i].z =  Fz;

		p_out.pos[i].x = (Fx * dt) + p_in.pos[i].x;
		p_out.pos[i].y = (Fy * dt) + p_in.pos[i].y;
		p_out.pos[i].z = (Fz * dt) + p_in.pos[i].z;

		//printf("%f %f %f %f %f %f %f %f xaxa\n", p_out.pos[i].x,p_out.pos[i].y,p_out.pos[i].z,p_out.pos[i].w,p_out.vel[i].x,p_out.vel[i].y,p_out.vel[i].z,p_out.vel[i].w);

	}
}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
	/* DOPLNTE: telo funkce pro nacitani castic */
	for (int i = 0; i < N; i++)
	{
		fscanf(fp, "%f %f %f %f %f %f %f \n",
				&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
				&p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
				&p.pos[i].w);
	}
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
	for (int i = 0; i < N; i++)
	{
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
				p.pos[i].x, p.pos[i].y, p.pos[i].z,
				p.vel[i].x, p.vel[i].y, p.vel[i].z,
				p.pos[i].w);
	}
	/* DOPLNTE: telo funkce pro ulozeni castic */
}
