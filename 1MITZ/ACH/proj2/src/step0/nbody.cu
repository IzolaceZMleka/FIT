#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xpavli78
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"
#include <iostream>
#include <random>
#include <exception>
#include <string>
#include <cstdio>

using namespace std;

__global__ void calculate_gravitation_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
	/* DOPLNTE: telo funkce pro vypocet gravitacni rychlosti */
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
		float r;
/*
		if(i == 1)
			printf("%f %f %f %f %f %f %f %f xaxa\n", p.pos[i].x,p.pos[i].y,p.pos[i].z,p.pos[i].w,p.vel[i].x,p.vel[i].y,p.vel[i].z,p.vel[i].w);
*/
		for (int j = 0; j < N; j++) {
			float dx = p.pos[j].x - p.pos[i].x;
			float dy = p.pos[j].y - p.pos[i].y;
			float dz = p.pos[j].z - p.pos[i].z;

			r = sqrtf(dx*dx + dy*dy + dz*dz);
			float r3 = r * r * r;

			float velocity = (G * dt * p.pos[j].w) / r3;

			Fx += (r > COLLISION_DISTANCE) ? dx * velocity : 0.0f;
			Fy += (r > COLLISION_DISTANCE) ? dy * velocity : 0.0f;
			Fz += (r > COLLISION_DISTANCE) ? dz * velocity : 0.0f;
		}

		tmp_vel.coords[i].x += Fx;
		tmp_vel.coords[i].y += Fy;
		tmp_vel.coords[i].z += Fz;

	}
}

__global__ void calculate_collision_velocity(t_particles p, t_velocities tmp_vel, int N, float dt)
{
	/* DOPLNTE: telo funkce pro vypocet kolizni rychlosti */
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
		float r;

		for (int j = 0; j < N; j++) {
			float dx = p.pos[i].x - p.pos[j].x;
			float dy = p.pos[i].y - p.pos[j].y;
			float dz = p.pos[i].z - p.pos[j].z;

			r = sqrtf(dx*dx + dy*dy + dz*dz);

			float dw = p.pos[i].w - p.pos[j].w;
			float totalw = p.pos[i].w + p.pos[j].w;

			Fx += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2 * p.pos[j].w * p.vel[j].x + dw * p.vel[i].x) / totalw) - p.vel[i].x : 0.0f;
			Fy += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2 * p.pos[j].w * p.vel[j].y + dw * p.vel[i].y) / totalw) - p.vel[i].y : 0.0f;
			Fz += (r > 0.0f && r < COLLISION_DISTANCE) ? ((2 * p.pos[j].w * p.vel[j].z + dw * p.vel[i].z) / totalw) - p.vel[i].z : 0.0f;
		}

		tmp_vel.coords[i].x += Fx;
		tmp_vel.coords[i].y += Fy;
		tmp_vel.coords[i].z += Fz;

	}

}

__global__ void update_particle(t_particles p, t_velocities tmp_vel, int N, float dt)
{
	/* DOPLNTE: telo funkce pro vypocet kolizni rychlosti */
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < N) {
		p.vel[i].x += tmp_vel.coords[i].x;
		p.vel[i].y += tmp_vel.coords[i].y;
		p.vel[i].z += tmp_vel.coords[i].z;

		p.pos[i].x += p.vel[i].x * dt;
		p.pos[i].y += p.vel[i].y * dt;
		p.pos[i].z += p.vel[i].z * dt;

		tmp_vel.coords[i].x = 0.0f;
		tmp_vel.coords[i].y = 0.0f;
		tmp_vel.coords[i].z = 0.0f;
	}


}

__host__ void particles_read(FILE *fp, t_particles &p, int N)
{
	/* DOPLNTE: telo funkce pro nacitani castic */
	for (int i = 0; i < N; i++)
	{
		fscanf(fp, "%f %f %f %f %f %f %f \n",
				&p.pos[i].x, &p.pos[i].y, &p.pos[i].z,
				&p.vel[i].x, &p.vel[i].y, &p.vel[i].z,
				&p.pos[i].w);
	}
}

__host__  void particles_write(FILE *fp, t_particles &p, int N)
{
	for (int i = 0; i < N; i++)
	{
		fprintf(fp, "%10.10f %10.10f %10.10f %10.10f %10.10f %10.10f %10.10f \n",
				p.pos[i].x, p.pos[i].y, p.pos[i].z,
				p.vel[i].x, p.vel[i].y, p.vel[i].z,
				p.pos[i].w);
	}
	/* DOPLNTE: telo funkce pro ulozeni castic */
}
