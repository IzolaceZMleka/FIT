#include "hip/hip_runtime.h"
/*
 * Architektura procesoru (ACH 2017)
 * Projekt c. 2 (cuda)
 * Login: xpavli78
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"

#define cudaCheckError() __cudaCheckError( __FILE__, __LINE__ )

int main(int argc, char **argv)
{
    FILE *fp;
    struct timeval t1, t2;
    int N;
    float dt;
    int steps;
    int thr_blc;

    // parametry
    if (argc != 7)
    {
        printf("Usage: nbody <N> <dt> <steps> <thr/blc> <input> <output>\n");
        exit(1);
    }
    N = atoi(argv[1]);
    dt = atof(argv[2]);
    steps = atoi(argv[3]);
    thr_blc = atoi(argv[4]);

    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);

    // alokace pameti na CPU
    t_particles particles_cpu;
    size_t partArr = N*sizeof(float4);
    
    hipHostAlloc(&particles_cpu.pos, partArr, hipHostMallocDefault);
    hipHostAlloc(&particles_cpu.vel, partArr, hipHostMallocDefault);
    

    /* DOPLNTE: alokaci pameti na CPU */

    // nacteni castic ze souboru
    fp = fopen(argv[5], "r");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[5]);
        exit(1);
    }
    particles_read(fp, particles_cpu, N);
    fclose(fp);

    t_particles particles_gpu;
    t_velocities tmp_velocities_gpu;

    /* DOPLNTE: alokaci pameti na GPU */
    hipMalloc(&particles_gpu.pos, partArr);
    hipMalloc(&particles_gpu.vel, partArr);
    
    hipMalloc(&tmp_velocities_gpu.coords, partArr);
    hipMemset(&tmp_velocities_gpu.coords, 0.0f, partArr);

    /* DOPLNTE: kopirovani pameti na GPU */
    
    hipMemcpy(particles_gpu.pos, particles_cpu.pos, partArr, hipMemcpyHostToDevice);
    hipMemcpy(particles_gpu.vel, particles_cpu.vel, partArr, hipMemcpyHostToDevice);
    

    /* DOPLNTE: vypocet velikosti vypocetniho gridu */
    size_t grid = (N + thr_blc - 1) / thr_blc;;
    
    
    // vypocet
    gettimeofday(&t1, 0);

    for (int s = 0; s < steps; ++s)
    {
    	calculate_gravitation_velocity<<<grid, thr_blc>>>(particles_gpu, tmp_velocities_gpu, N, dt);
    	calculate_collision_velocity<<<grid, thr_blc>>>(particles_gpu, tmp_velocities_gpu, N, dt);
    	update_particle<<<grid, thr_blc>>>(particles_gpu, tmp_velocities_gpu, N, dt);
        /* DOPLNTE: volani vypocetnich kernelu */
    }
    hipDeviceSynchronize();
    gettimeofday(&t2, 0);

    // cas
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    /* DOPLNTE: koopirovani castic spatky na CPU */
    hipMemcpy(particles_cpu.pos, particles_gpu.pos, partArr, hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel, particles_gpu.vel, partArr, hipMemcpyDeviceToHost);

    // ulozeni castic do souboru
    fp = fopen(argv[6], "w");
    if (fp == NULL)
    {
        printf("Can't open file %s!\n", argv[6]);
        exit(1);
    }
    particles_write(fp, particles_cpu, N);
    fclose(fp);


    if(particles_cpu.pos)  hipFree(particles_cpu.pos);
    if(particles_cpu.vel)  hipFree(particles_cpu.vel);
    if(particles_gpu.pos)  hipFree(particles_gpu.pos);
    if(particles_gpu.vel)  hipFree(particles_gpu.vel);
    if(tmp_velocities_gpu.coords)  hipFree(tmp_velocities_gpu.coords);


    return 0;
}

inline void __cudaCheckError( const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
  {
	  cerr<<"cudaCheckError() failed at "<<file<<":"<<line<<" : "<<hipGetErrorString( err )<<endl;
    exit( -1 );
  }
}
